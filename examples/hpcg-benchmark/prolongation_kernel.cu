#include "hip/hip_runtime.h"
#include <cstdlib>

#include <ginkgo/ginkgo.hpp>


#define INSTANTIATE_FOR_EACH_VALUE_TYPE(_macro) \
    template _macro(float);                     \
    template _macro(double);

#define PROLONGATION_KERNEL(_type)                                           \
    void prolongation_kernel(int nx, int ny, int nz, const _type* coeffs,    \
                             const _type* rhs, const int rhs_size, _type* x, \
                             const int x_size);

namespace {

// geo is coarse
template <typename ValueType>
__global__ void prolongation_kernel_impl(int nx, int ny, int nz,
                                         const ValueType* coeffs,
                                         const ValueType* coarse_rhs,
                                         const int rhs_size, ValueType* fine_x,
                                         const int x_size)
{
    const auto nt_x = blockDim.x;
    const auto f_x = threadIdx.x + nt_x * blockIdx.x;
    const auto f_y = blockIdx.y;
    const auto f_z = blockIdx.z;

    const auto f_x_on_coarse = f_x % 2;
    const auto f_y_on_coarse = f_y % 2;
    const auto f_z_on_coarse = f_z % 2;

    const auto f_id =
        f_z * (2 * nx + 1) * (2 * ny + 1) + f_y * (2 * nx + 1) + f_x;
    const auto c_id = ((f_z - f_z_on_coarse) / 2) * (nx + 1) * (ny + 1) +
                      ((f_y - f_y_on_coarse) / 2) * (nx + 1) +
                      ((f_x - f_x_on_coarse) / 2);

    if (f_x <= 2 * nx) {
        if (!f_z_on_coarse) {
            if (!f_y_on_coarse) {
                if (!f_x_on_coarse) {
                    // const auto c_id = (f_z / 2) * (nx + 1) * (ny + 1) +
                    //                   (f_y / 2) * (nx + 1) + (f_x / 2);
                    fine_x[f_id] =
                        coeffs[1] * coeffs[1] * coeffs[1] * coarse_rhs[c_id];
                } else {
                    // const auto c_id = (f_z / 2) * (nx + 1) * (ny + 1) +
                    //                   (f_y / 2) * (nx + 1) + ((f_x - 1) /
                    // 2);
                    fine_x[f_id] = coeffs[1] * coeffs[1] * coeffs[0] *
                                   (coarse_rhs[c_id] + coarse_rhs[c_id + 1]);
                }
            } else {
                if (!f_x_on_coarse) {
                    // const auto c_id = (f_z / 2) * (nx + 1) * (ny + 1) +
                    //                   ((f_y - 1) / 2) * (nx + 1) + (f_x /
                    // 2);
                    fine_x[f_id] =
                        coeffs[1] * coeffs[1] * coeffs[0] *
                        (coarse_rhs[c_id] + coarse_rhs[c_id + nx + 1]);
                } else {
                    // const auto c_id = (f_z / 2) * (nx + 1) * (ny + 1) +
                    //                   ((f_y - 1) / 2) * (nx + 1) +
                    //                   ((f_x - 1) / 2);
                    fine_x[f_id] =
                        coeffs[1] * coeffs[0] * coeffs[0] *
                        (coarse_rhs[c_id] + coarse_rhs[c_id + nx + 1] +
                         coarse_rhs[c_id + 1] + coarse_rhs[c_id + 2 + nx]);
                }
            }
        } else {
            if (!f_y_on_coarse) {
                if (!f_x_on_coarse) {
                    // const auto c_id = ((f_z - 1) / 2) * (nx + 1) * (ny +
                    // 1) +
                    //                   (f_y / 2) * (nx + 1) + (f_x / 2);
                    fine_x[f_id] = coeffs[1] * coeffs[1] * coeffs[0] *
                                   (coarse_rhs[c_id] +
                                    coarse_rhs[c_id + (nx + 1) * (ny + 1)]);
                } else {
                    // const auto c_id = ((f_z - 1) / 2) * (nx + 1) * (ny +
                    // 1) +
                    //                   (f_y / 2) * (nx + 1) + ((f_x - 1) /
                    // 2);
                    fine_x[f_id] = coeffs[1] * coeffs[0] * coeffs[0] *
                                   (coarse_rhs[c_id] +
                                    coarse_rhs[c_id + (nx + 1) * (ny + 1)] +
                                    coarse_rhs[c_id + 1] +
                                    coarse_rhs[c_id + (nx + 1) * (ny + 1) + 1]);
                }
            } else {
                if (!f_x_on_coarse) {
                    // const auto c_id = ((f_z - 1) / 2) * (nx + 1) * (ny +
                    // 1) +
                    //                   ((f_y - 1) / 2) * (nx + 1) + (f_x /
                    // 2);
                    fine_x[f_id] =
                        coeffs[1] * coeffs[0] * coeffs[0] *
                        (coarse_rhs[c_id] +
                         coarse_rhs[c_id + (nx + 1) * (ny + 1)] +
                         coarse_rhs[c_id + (nx + 1)] +
                         coarse_rhs[c_id + (nx + 1) * (ny + 1) + (nx + 1)]);
                } else {
                    // const auto c_id = ((f_z - 1) / 2) * (nx + 1) * (ny +
                    // 1) +
                    //                   ((f_y - 1) / 2) * (nx + 1) +
                    //                   ((f_x - 1) / 2);
                    fine_x[f_id] =
                        coeffs[0] * coeffs[0] * coeffs[0] *
                        (coarse_rhs[c_id] + coarse_rhs[c_id + 1] +
                         coarse_rhs[c_id + (nx + 1)] +
                         coarse_rhs[c_id + (nx + 1) * (ny + 1)] +
                         coarse_rhs[c_id + (nx + 1) + 1] +
                         coarse_rhs[c_id + (nx + 1) * (ny + 1) + 1] +
                         coarse_rhs[c_id + (nx + 1) + (nx + 1) * (ny + 1)] +
                         coarse_rhs[c_id + (nx + 1) + (nx + 1) * (ny + 1) + 1]);
                }
            }
        }


        // for (auto ofs_z = -1; ofs_z < 2; ofs_z++) {
        //     if (f_z + ofs_z >= 0 && f_z + ofs_z <= 2 * nz) {
        //         for (auto ofs_y = -1; ofs_y < 2; ofs_y++) {
        //             if (f_y + ofs_y >= 0 && f_y + ofs_y <= 2 * ny) {
        //                 for (auto ofs_x = -1; ofs_x < 2; ofs_x++) {
        //                     if (f_x + ofs_x >= 0 && f_x + ofs_x <= 2 * nx) {
        //                         auto f_offset =
        //                             ofs_z * (2 * nx + 1) * (2 * ny + 1) +
        //                             ofs_y * (2 * nx + 1) + ofs_x;
        //                         fine_x[f_id + f_offset] +=
        //                             coeffs[ofs_z + 1] * coeffs[ofs_y + 1] *
        //                             coeffs[ofs_x + 1] * coarse_rhs[c_id];

        //                         // __syncthreads();
        //                     }
        //                 }
        //             }
        //         }
        //     }
        // }
    }
}
}  // namespace

template <typename ValueType>
void prolongation_kernel(int nx, int ny, int nz, const ValueType* coeffs,
                         const ValueType* rhs, const int rhs_size, ValueType* x,
                         const int x_size)
{
    constexpr int block_size = 32;
    const auto grid_size =
        dim3((2 * nx + block_size - 1) / block_size, 2 * ny + 1, 2 * nz + 1);
    prolongation_kernel_impl<<<grid_size, block_size>>>(nx, ny, nz, coeffs, rhs,
                                                        rhs_size, x, x_size);
}

INSTANTIATE_FOR_EACH_VALUE_TYPE(PROLONGATION_KERNEL);