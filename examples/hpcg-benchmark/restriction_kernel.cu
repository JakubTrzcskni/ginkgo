#include "hip/hip_runtime.h"
#include <cstdlib>

#include <ginkgo/ginkgo.hpp>


#define INSTANTIATE_FOR_EACH_VALUE_TYPE(_macro) \
    template _macro(float);                     \
    template _macro(double);

#define RESTRICTION_KERNEL(_type) void restriction_kernel(_type* x);

namespace {
template <typename ValueType>
__global__ void restriction_kernel_impl()
{}
}  // namespace

template <typename ValueType>
void restriction_kernel(ValueType* x)
{
    // constexpr int block_size = 512;
    // const auto grid_size = (size + block_size - 1) / block_size;
}

INSTANTIATE_FOR_EACH_VALUE_TYPE(RESTRICTION_KERNEL);