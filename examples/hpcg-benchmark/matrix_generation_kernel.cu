#include "hip/hip_runtime.h"
#include <cstdlib>

#include <ginkgo/ginkgo.hpp>


#define INSTANTIATE_FOR_EACH_VALUE_TYPE(_macro) \
    template _macro(float);                     \
    template _macro(double);

#define MATRIX_GENERATION_KERNEL(_type) void matrix_generation_kernel(_type* x);

namespace {
template <typename ValueType>
__global__ void matrix_generation_kernel_impl()
{}
}  // namespace

template <typename ValueType>
void matrix_generation_kernel(ValueType* x)
{
    // constexpr int block_size = 512;
    // const auto grid_size = (size + block_size - 1) / block_size;
}

INSTANTIATE_FOR_EACH_VALUE_TYPE(MATRIX_GENERATION_KERNEL);